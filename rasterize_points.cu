/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>
#include <torch/extension.h>
#include <cstdio>
#include <functional>
#include <tuple>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
  auto lambda = [&t](size_t N) {
    t.resize_({(long long)N});
    return reinterpret_cast<char*>(t.contiguous().data_ptr());
  };
  return lambda;
}

std::tuple<int,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor>
RasterizeGaussiansCUDA(const torch::Tensor& background,
                       const torch::Tensor& means3D,
                       const torch::Tensor& colors,
                       const torch::Tensor& opacity,
                       const torch::Tensor& scales,
                       const torch::Tensor& rotations,
                       const float scale_modifier,
                       const torch::Tensor& cov3D_precomp,
                       const torch::Tensor& viewmatrix,
                       const torch::Tensor& projmatrix,
                       const torch::Tensor& projmatrix_raw,
                       const float tan_fovx,
                       const float tan_fovy,
                       const int image_height,
                       const int image_width,
                       const torch::Tensor& sh,
                       const int degree,
                       const torch::Tensor& campos,
                       const bool prefiltered,
                       const bool debug) {
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }

  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii =
      torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Tensor n_touched =
      torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Tensor out_depth = torch::full({1, H, W}, 0.0, float_opts);
  torch::Tensor out_opaticy = torch::full({1, H, W}, 0.0, float_opts);

  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);

  int rendered = 0;
  if (P != 0) {
    int M = 0;
    if (sh.size(0) != 0) {
      M = sh.size(1);
    }

    rendered = CudaRasterizer::Rasterizer::forward(
        geomFunc,
        binningFunc,
        imgFunc,
        P,
        degree,
        M,
        background.contiguous().data_ptr<float>(),
        W,
        H,
        means3D.contiguous().data_ptr<float>(),
        sh.contiguous().data_ptr<float>(),
        colors.contiguous().data_ptr<float>(),
        opacity.contiguous().data_ptr<float>(),
        scales.contiguous().data_ptr<float>(),
        scale_modifier,
        rotations.contiguous().data_ptr<float>(),
        cov3D_precomp.contiguous().data_ptr<float>(),
        viewmatrix.contiguous().data_ptr<float>(),
        projmatrix.contiguous().data_ptr<float>(),
        campos.contiguous().data_ptr<float>(),
        tan_fovx,
        tan_fovy,
        prefiltered,
        out_color.contiguous().data_ptr<float>(),
        out_depth.contiguous().data_ptr<float>(),
        out_opaticy.contiguous().data_ptr<float>(),
        radii.contiguous().data_ptr<int>(),
        n_touched.contiguous().data_ptr<int>(),
        debug);
  }
  return std::make_tuple(rendered,
                         out_color,
                         radii,
                         geomBuffer,
                         binningBuffer,
                         imgBuffer,
                         out_depth,
                         out_opaticy,
                         n_touched);
}

std::tuple<int,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor>
RasterizeSemanticGaussiansCUDA(const torch::Tensor& background_color,
                               const torch::Tensor& background_semantics,
                               const torch::Tensor& means3D,
                               const torch::Tensor& colors,
                               const torch::Tensor& semantics,
                               const torch::Tensor& opacity,
                               const torch::Tensor& scales,
                               const torch::Tensor& rotations,
                               const float scale_modifier,
                               const torch::Tensor& cov3D_precomp,
                               const torch::Tensor& viewmatrix,
                               const torch::Tensor& projmatrix,
                               const torch::Tensor& projmatrix_raw,
                               const float tan_fovx,
                               const float tan_fovy,
                               const int image_height,
                               const int image_width,
                               const torch::Tensor& sh,
                               const torch::Tensor& semantic_sh,
                               const int degree,
                               const torch::Tensor& campos,
                               const bool prefiltered,
                               const bool debug) {
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }

  const int P = means3D.size(0);  // the number of 3D Gaussian ellipsoids
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_colors = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  // ?
  torch::Tensor out_semantics =
      torch::full({NUM_SEMANTIC_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii =
      torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Tensor n_touched =
      torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Tensor out_depth = torch::full({1, H, W}, 0.0, float_opts);
  torch::Tensor out_opaticy = torch::full({1, H, W}, 0.0, float_opts);

  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor semantic_geometry_buffer =
      torch::empty({0}, options.device(device));
  torch::Tensor binning_buffer = torch::empty({0}, options.device(device));
  torch::Tensor img_buffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> semantic_geometry_func =
      resizeFunctional(semantic_geometry_buffer);
  std::function<char*(size_t)> binning_func = resizeFunctional(binning_buffer);
  std::function<char*(size_t)> img_func = resizeFunctional(img_buffer);

  int num_rendered = 0;
  if (P != 0) {
    int M = 0;
    if (sh.size(0) != 0) {
      M = sh.size(1);
    }
    int semantic_M = 0;
    if (semantic_sh.size(0) != 0) {
      semantic_M = semantic_sh.size(1);
    }

    num_rendered = CudaRasterizer::SemanticRasterizer::forward(
        semantic_geometry_func,
        binning_func,
        img_func,
        P,
        degree,
        M,
        semantic_M,
        background_color.contiguous().data_ptr<float>(),
        background_semantics.contiguous().data_ptr<float>(),
        W,
        H,
        means3D.contiguous().data_ptr<float>(),
        sh.contiguous().data_ptr<float>(),
        semantic_sh.contiguous().data_ptr<float>(),
        colors.contiguous().data_ptr<float>(),
        semantics.contiguous().data_ptr<float>(),
        opacity.contiguous().data_ptr<float>(),
        scales.contiguous().data_ptr<float>(),
        scale_modifier,
        rotations.contiguous().data_ptr<float>(),
        cov3D_precomp.contiguous().data_ptr<float>(),
        viewmatrix.contiguous().data_ptr<float>(),
        projmatrix.contiguous().data_ptr<float>(),
        campos.contiguous().data_ptr<float>(),
        tan_fovx,
        tan_fovy,
        prefiltered,
        out_colors.contiguous().data_ptr<float>(),
        out_semantics.contiguous().data_ptr<float>(),
        out_depth.contiguous().data_ptr<float>(),
        out_opaticy.contiguous().data_ptr<float>(),
        radii.contiguous().data_ptr<int>(),
        n_touched.contiguous().data_ptr<int>(),
        debug);
  }
  return std::make_tuple(num_rendered,
                         out_colors,
                         out_semantics,
                         radii,
                         semantic_geometry_buffer,
                         binning_buffer,
                         img_buffer,
                         out_depth,
                         out_opaticy,
                         n_touched);
}

std::tuple<torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor>
RasterizeGaussiansBackwardCUDA(const torch::Tensor& background,
                               const torch::Tensor& means3D,
                               const torch::Tensor& radii,
                               const torch::Tensor& colors,
                               const torch::Tensor& scales,
                               const torch::Tensor& rotations,
                               const float scale_modifier,
                               const torch::Tensor& cov3D_precomp,
                               const torch::Tensor& viewmatrix,
                               const torch::Tensor& projmatrix,
                               const torch::Tensor& projmatrix_raw,
                               const float tan_fovx,
                               const float tan_fovy,
                               const torch::Tensor& dL_dout_color,
                               const torch::Tensor& dL_dout_depths,
                               const torch::Tensor& sh,
                               const int degree,
                               const torch::Tensor& campos,
                               const torch::Tensor& geomBuffer,
                               const int R,
                               const torch::Tensor& binningBuffer,
                               const torch::Tensor& imageBuffer,
                               const bool debug) {
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);

  int M = 0;
  if (sh.size(0) != 0) {
    M = sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_ddepths = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  torch::Tensor dL_dtau = torch::zeros({P, 6}, means3D.options());

  if (P != 0) {
    CudaRasterizer::Rasterizer::backward(
        P,
        degree,
        M,
        R,
        background.contiguous().data_ptr<float>(),
        W,
        H,
        means3D.contiguous().data_ptr<float>(),
        sh.contiguous().data_ptr<float>(),
        colors.contiguous().data_ptr<float>(),
        scales.data_ptr<float>(),
        scale_modifier,
        rotations.data_ptr<float>(),
        cov3D_precomp.contiguous().data_ptr<float>(),
        viewmatrix.contiguous().data_ptr<float>(),
        projmatrix.contiguous().data_ptr<float>(),
        projmatrix_raw.contiguous().data_ptr<float>(),
        campos.contiguous().data_ptr<float>(),
        tan_fovx,
        tan_fovy,
        radii.contiguous().data_ptr<int>(),
        reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
        reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
        reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
        dL_dout_color.contiguous().data_ptr<float>(),
        dL_dout_depths.contiguous().data_ptr<float>(),
        dL_dmeans2D.contiguous().data_ptr<float>(),
        dL_dconic.contiguous().data_ptr<float>(),
        dL_dopacity.contiguous().data_ptr<float>(),
        dL_dcolors.contiguous().data_ptr<float>(),
        dL_ddepths.contiguous().data_ptr<float>(),
        dL_dmeans3D.contiguous().data_ptr<float>(),
        dL_dcov3D.contiguous().data_ptr<float>(),
        dL_dsh.contiguous().data_ptr<float>(),
        dL_dscales.contiguous().data_ptr<float>(),
        dL_drotations.contiguous().data_ptr<float>(),
        dL_dtau.contiguous().data_ptr<float>(),
        debug);
  }

  return std::make_tuple(dL_dmeans2D,
                         dL_dcolors,
                         dL_dopacity,
                         dL_dmeans3D,
                         dL_dcov3D,
                         dL_dsh,
                         dL_dscales,
                         dL_drotations,
                         dL_dtau);
}

std::tuple<torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor,
           torch::Tensor>
RasterizeSemanticGaussiansBackwardCUDA(
    const torch::Tensor& background_color,
    const torch::Tensor& background_semantics,
    const torch::Tensor& means3D,
    const torch::Tensor& radii,
    const torch::Tensor& colors,
    const torch::Tensor& semantics,
    const torch::Tensor& scales,
    const torch::Tensor& rotations,
    const float scale_modifier,
    const torch::Tensor& cov3D_precomp,
    const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
    const torch::Tensor& projmatrix_raw,
    const float tan_fovx,
    const float tan_fovy,
    const torch::Tensor& dL_dout_color,
    const torch::Tensor& dL_dout_semantics,
    const torch::Tensor& dL_dout_depth,
    const torch::Tensor& sh,
    const torch::Tensor& semantic_sh,
    const int degree,
    const torch::Tensor& campos,
    const torch::Tensor& semantic_geometry_buffer,
    const int R,
    const torch::Tensor& binning_buffer,
    const torch::Tensor& image_buffer,
    const bool debug) {
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);

  int M = 0;
  if (sh.size(0) != 0) {
    M = sh.size(1);
  }
  int semantic_M = 0;
  if (semantic_sh.size(0) != 0) {
    semantic_M = sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  // ?
  torch::Tensor dL_dsemantics =
      torch::zeros({P, NUM_SEMANTIC_CHANNELS}, means3D.options());
  torch::Tensor dL_ddepths = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dsemantic_sh = torch::zeros({P, M, NUM_SEMANTIC_CHANNELS}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  torch::Tensor dL_dtau = torch::zeros({P, 6}, means3D.options());

  if (P != 0) {
    CudaRasterizer::SemanticRasterizer::backward(
        P,
        degree,
        M,
        semantic_M,
        R,
        background_color.contiguous().data_ptr<float>(),
        background_semantics.contiguous().data_ptr<float>(),
        W,
        H,
        means3D.contiguous().data_ptr<float>(),
        sh.contiguous().data_ptr<float>(),
        semantic_sh.contiguous().data_ptr<float>(),
        colors.contiguous().data_ptr<float>(),
        semantics.contiguous().data_ptr<float>(),
        scales.data_ptr<float>(),
        scale_modifier,
        rotations.data_ptr<float>(),
        cov3D_precomp.contiguous().data_ptr<float>(),
        viewmatrix.contiguous().data_ptr<float>(),
        projmatrix.contiguous().data_ptr<float>(),
        projmatrix_raw.contiguous().data_ptr<float>(),
        campos.contiguous().data_ptr<float>(),
        tan_fovx,
        tan_fovy,
        radii.contiguous().data_ptr<int>(),
        reinterpret_cast<char*>(
            semantic_geometry_buffer.contiguous().data_ptr()),
        reinterpret_cast<char*>(binning_buffer.contiguous().data_ptr()),
        reinterpret_cast<char*>(image_buffer.contiguous().data_ptr()),
        dL_dout_color.contiguous().data_ptr<float>(),
        dL_dout_semantics.contiguous().data_ptr<float>(),
        dL_dout_depth.contiguous().data_ptr<float>(),
        dL_dmeans2D.contiguous().data_ptr<float>(),
        dL_dconic.contiguous().data_ptr<float>(),
        dL_dopacity.contiguous().data_ptr<float>(),
        dL_dcolors.contiguous().data_ptr<float>(),
        dL_dsemantics.contiguous().data_ptr<float>(),
        dL_ddepths.contiguous().data_ptr<float>(),
        dL_dmeans3D.contiguous().data_ptr<float>(),
        dL_dcov3D.contiguous().data_ptr<float>(),
        dL_dsh.contiguous().data_ptr<float>(),
        dL_dsemantic_sh.contiguous().data_ptr<float>(),
        dL_dscales.contiguous().data_ptr<float>(),
        dL_drotations.contiguous().data_ptr<float>(),
        dL_dtau.contiguous().data_ptr<float>(),
        debug);
  }

  return std::make_tuple(dL_dmeans2D,
                         dL_dcolors,
                         dL_dsemantics,
                         dL_dopacity,
                         dL_dmeans3D,
                         dL_dcov3D,
                         dL_dsh,
                         dL_dsemantic_sh,
                         dL_dscales,
                         dL_drotations,
                         dL_dtau);
}
torch::Tensor markVisible(torch::Tensor& means3D,
                          torch::Tensor& viewmatrix,
                          torch::Tensor& projmatrix) {
  const int P = means3D.size(0);

  torch::Tensor present =
      torch::full({P}, false, means3D.options().dtype(at::kBool));

  if (P != 0) {
    CudaRasterizer::Rasterizer::markVisible(
        P,
        means3D.contiguous().data_ptr<float>(),
        viewmatrix.contiguous().data_ptr<float>(),
        projmatrix.contiguous().data_ptr<float>(),
        present.contiguous().data_ptr<bool>());
  }

  return present;
}
